#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * Copyright (C) 2025 Intel Corporation, All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "cutlass_unit_test.h"

#include <iostream>
#include <iomanip>
#include <utility>
#include <type_traits>
#include <vector>
#include <numeric>

#include <cute/tensor.hpp>

using namespace cute;

#ifdef CUTLASS_ENABLE_SYCL
namespace sc = compat;
namespace sc_exp = compat::experimental;
namespace sycl_ext = sycl::ext::oneapi::experimental;

CUTLASS_GLOBAL void
test(double const* g_in, double* g_out, sycl::local_ptr<char> base_smem)
{
  auto smem = reinterpret_cast<double*>((char*)base_smem);
  smem[ThreadIdxX()] = g_in[ThreadIdxX()];

  syncthreads();

  g_out[ThreadIdxX()] = 2 * smem[ThreadIdxX()];
}

CUTLASS_GLOBAL void
test2(double const* g_in, double* g_out, sycl::local_ptr<char> base_smem)
{
  using namespace cute;

  auto smem = reinterpret_cast<double*>((char*)base_smem);

  auto s_tensor = make_tensor(make_smem_ptr(smem + ThreadIdxX()), Int<1>{});
  auto g_tensor = make_tensor(make_gmem_ptr(g_in + ThreadIdxX()), Int<1>{});

  copy(g_tensor, s_tensor);

  syncthreads();

  g_out[ThreadIdxX()] = 2 * smem[ThreadIdxX()];
}

#else

__global__ void
test(double const* g_in, double* g_out)
{
  extern __shared__ double smem[];

  smem[threadIdx.x] = g_in[threadIdx.x];

  __syncthreads();

  g_out[threadIdx.x] = 2 * smem[threadIdx.x];
}

__global__ void
test2(double const* g_in, double* g_out)
{
  using namespace cute;

  extern __shared__ double smem[];

  auto s_tensor = make_tensor(make_smem_ptr(smem + threadIdx.x), Int<1>{});
  auto g_tensor = make_tensor(make_gmem_ptr(g_in + threadIdx.x), Int<1>{});

  copy(g_tensor, s_tensor);

  __syncthreads();

  g_out[threadIdx.x] = 2 * smem[threadIdx.x];
}

#endif

TEST(SM80_CuTe_Ampere, CpSync)
{
  constexpr int count = 32;
  host_vector<double> h_in(count);
  for (int i = 0; i < count; ++i) {
    h_in[i] = double(i);
  }

  device_vector<double> d_in(h_in);

  device_vector<double> d_out(count, -1);
  #if defined(CUTLASS_ENABLE_SYCL)
    sc_exp::launch<test>(sc_exp::launch_policy{sc::dim3(1), sc::dim3(count),
              sc_exp::local_mem_size{sizeof(double) * count}},
              d_in.data(), d_out.data());
    sc::wait_and_throw();
  #else
    test<<<1, count, sizeof(double) * count>>>(
      thrust::raw_pointer_cast(d_in.data()),
      thrust::raw_pointer_cast(d_out.data()));
  #endif
  host_vector<double> h_result = d_out;

  device_vector<double> d_out_cp_async(count, -2);
  #if defined(CUTLASS_ENABLE_SYCL)
    sc_exp::launch<test2>(sc_exp::launch_policy{sc::dim3(1), sc::dim3(count),
              sc_exp::local_mem_size{sizeof(double) * count}},
              d_in.data(), d_out_cp_async.data());
    sc::wait_and_throw();
  #else
    test2<<<1, count, sizeof(double) * count>>>(
      thrust::raw_pointer_cast(d_in.data()),
      thrust::raw_pointer_cast(d_out_cp_async.data()));
  #endif
  host_vector<double> h_result_cp_async = d_out_cp_async;

  for (int i = 0; i < count; ++i) {
    EXPECT_EQ(h_result[i], h_result_cp_async[i]);
  }
}
